#include "hip/hip_runtime.h"
//***********************************************************************************
// 2018.04.01 created by Zexlus1126
//
//    Example 002
// This is a simple demonstration on calculating merkle root from merkle branch 
// and solving a block (#286819) which the information is downloaded from Block Explorer 
//***********************************************************************************

#include <iostream>
#include <fstream>
#include <string>
#include <chrono>

#include <cstdio>
#include <cstring>

#include <cassert>
#include <hip/hip_runtime.h>

// #include "sha256.h"

#define BLOCK_SIZE 128

#define _rotl(v, s) ((v)<<(s) | (v)>>(32-(s)))
#define _rotr(v, s) ((v)>>(s) | (v)<<(32-(s)))

#define _swap(x, y) (((x)^=(y)), ((y)^=(x)), ((x)^=(y)))

typedef unsigned int WORD;
typedef unsigned char BYTE;

typedef union _sha256_ctx{
	WORD h[8];
	BYTE b[32];
}SHA256;

////////////////////////   Block   /////////////////////

typedef struct _block
{
    unsigned int version;
    unsigned char prevhash[32];
    unsigned char merkle_root[32];
    unsigned int ntime;
    unsigned int nbits;
    unsigned int nonce;
}HashBlock;

typedef struct _sharedData
{
    HashBlock block;
    SHA256 sha256_ctx;
    SHA256 tmp;
}SharedData;

__constant__ static const WORD k[64] = {
	0x428a2f98,0x71374491,0xb5c0fbcf,0xe9b5dba5,0x3956c25b,0x59f111f1,0x923f82a4,0xab1c5ed5,
	0xd807aa98,0x12835b01,0x243185be,0x550c7dc3,0x72be5d74,0x80deb1fe,0x9bdc06a7,0xc19bf174,
	0xe49b69c1,0xefbe4786,0x0fc19dc6,0x240ca1cc,0x2de92c6f,0x4a7484aa,0x5cb0a9dc,0x76f988da,
	0x983e5152,0xa831c66d,0xb00327c8,0xbf597fc7,0xc6e00bf3,0xd5a79147,0x06ca6351,0x14292967,
	0x27b70a85,0x2e1b2138,0x4d2c6dfc,0x53380d13,0x650a7354,0x766a0abb,0x81c2c92e,0x92722c85,
	0xa2bfe8a1,0xa81a664b,0xc24b8b70,0xc76c51a3,0xd192e819,0xd6990624,0xf40e3585,0x106aa070,
	0x19a4c116,0x1e376c08,0x2748774c,0x34b0bcb5,0x391c0cb3,0x4ed8aa4a,0x5b9cca4f,0x682e6ff3,
	0x748f82ee,0x78a5636f,0x84c87814,0x8cc70208,0x90befffa,0xa4506ceb,0xbef9a3f7,0xc67178f2
};
static const WORD h_k[64] = {
	0x428a2f98,0x71374491,0xb5c0fbcf,0xe9b5dba5,0x3956c25b,0x59f111f1,0x923f82a4,0xab1c5ed5,
	0xd807aa98,0x12835b01,0x243185be,0x550c7dc3,0x72be5d74,0x80deb1fe,0x9bdc06a7,0xc19bf174,
	0xe49b69c1,0xefbe4786,0x0fc19dc6,0x240ca1cc,0x2de92c6f,0x4a7484aa,0x5cb0a9dc,0x76f988da,
	0x983e5152,0xa831c66d,0xb00327c8,0xbf597fc7,0xc6e00bf3,0xd5a79147,0x06ca6351,0x14292967,
	0x27b70a85,0x2e1b2138,0x4d2c6dfc,0x53380d13,0x650a7354,0x766a0abb,0x81c2c92e,0x92722c85,
	0xa2bfe8a1,0xa81a664b,0xc24b8b70,0xc76c51a3,0xd192e819,0xd6990624,0xf40e3585,0x106aa070,
	0x19a4c116,0x1e376c08,0x2748774c,0x34b0bcb5,0x391c0cb3,0x4ed8aa4a,0x5b9cca4f,0x682e6ff3,
	0x748f82ee,0x78a5636f,0x84c87814,0x8cc70208,0x90befffa,0xa4506ceb,0xbef9a3f7,0xc67178f2
};

__device__ __forceinline__ void sha256_transform(SHA256 *ctx, const BYTE *msg)
{
	WORD a, b, c, d, e, f, g, h;
	WORD i, j;
	
	// Create a 64-entry message schedule array w[0..63] of 32-bit words
	WORD w[64];
	// Copy chunk into first 16 words w[0..15] of the message schedule array
	w[0]  = (msg[0]<<24)  | (msg[1]<<16)  | (msg[2]<<8)  | msg[3];
    w[1]  = (msg[4]<<24)  | (msg[5]<<16)  | (msg[6]<<8)  | msg[7];
    w[2]  = (msg[8]<<24)  | (msg[9]<<16)  | (msg[10]<<8) | msg[11];
    w[3]  = (msg[12]<<24) | (msg[13]<<16) | (msg[14]<<8) | msg[15];
    w[4]  = (msg[16]<<24) | (msg[17]<<16) | (msg[18]<<8) | msg[19];
    w[5]  = (msg[20]<<24) | (msg[21]<<16) | (msg[22]<<8) | msg[23];
    w[6]  = (msg[24]<<24) | (msg[25]<<16) | (msg[26]<<8) | msg[27];
    w[7]  = (msg[28]<<24) | (msg[29]<<16) | (msg[30]<<8) | msg[31];
    w[8]  = (msg[32]<<24) | (msg[33]<<16) | (msg[34]<<8) | msg[35];
    w[9]  = (msg[36]<<24) | (msg[37]<<16) | (msg[38]<<8) | msg[39];
    w[10] = (msg[40]<<24) | (msg[41]<<16) | (msg[42]<<8) | msg[43];
    w[11] = (msg[44]<<24) | (msg[45]<<16) | (msg[46]<<8) | msg[47];
    w[12] = (msg[48]<<24) | (msg[49]<<16) | (msg[50]<<8) | msg[51];
    w[13] = (msg[52]<<24) | (msg[53]<<16) | (msg[54]<<8) | msg[55];
    w[14] = (msg[56]<<24) | (msg[57]<<16) | (msg[58]<<8) | msg[59];
    w[15] = (msg[60]<<24) | (msg[61]<<16) | (msg[62]<<8) | msg[63];
	
	// Extend the first 16 words into the remaining 48 words w[16..63] of the message schedule array:
	for(i=16;i<64;++i)
	{
		WORD s0 = (_rotr(w[i-15], 7)) ^ (_rotr(w[i-15], 18)) ^ (w[i-15]>>3);
		WORD s1 = (_rotr(w[i-2], 17)) ^ (_rotr(w[i-2], 19))  ^ (w[i-2]>>10);
		w[i] = w[i-16] + s0 + w[i-7] + s1;
	}
	
	
	// Initialize working variables to current hash value
	a = ctx->h[0];
	b = ctx->h[1];
	c = ctx->h[2];
	d = ctx->h[3];
	e = ctx->h[4];
	f = ctx->h[5];
	g = ctx->h[6];
	h = ctx->h[7];
	
	// Compress function main loop:
	for(i=0;i<64;++i)
	{
		WORD S0 = (_rotr(a, 2)) ^ (_rotr(a, 13)) ^ (_rotr(a, 22));
		WORD S1 = (_rotr(e, 6)) ^ (_rotr(e, 11)) ^ (_rotr(e, 25));
		WORD ch = (e & f) ^ ((~e) & g);
		WORD maj = (a & b) ^ (a & c) ^ (b & c);
		WORD temp1 = h + S1 + ch + k[i] + w[i];
		WORD temp2 = S0 + maj;
		
		h = g;
		g = f;
		f = e;
		e = d + temp1;
		d = c;
		c = b;
		b = a;
		a = temp1 + temp2;
	}
	
	// Add the compressed chunk to the current hash value
	ctx->h[0] += a;
	ctx->h[1] += b;
	ctx->h[2] += c;
	ctx->h[3] += d;
	ctx->h[4] += e;
	ctx->h[5] += f;
	ctx->h[6] += g;
	ctx->h[7] += h;
	
}

__device__ void sha256(SHA256 *ctx, const BYTE *msg, size_t len)
{
	// Initialize hash values:
	// (first 32 bits of the fractional parts of the square roots of the first 8 primes 2..19):
	ctx->h[0] = 0x6a09e667;
	ctx->h[1] = 0xbb67ae85;
	ctx->h[2] = 0x3c6ef372;
	ctx->h[3] = 0xa54ff53a;
	ctx->h[4] = 0x510e527f;
	ctx->h[5] = 0x9b05688c;
	ctx->h[6] = 0x1f83d9ab;
	ctx->h[7] = 0x5be0cd19;
	
	
	WORD i, j;
	size_t remain = len % 64;
	size_t total_len = len - remain;
	
	// Process the message in successive 512-bit chunks
	// For each chunk:
	for(i=0;i<total_len;i+=64)
	{
		sha256_transform(ctx, &msg[i]);
	}
	
	// Process remain data
	BYTE m[64] = {};
	for(i=total_len, j=0;i<len;++i, ++j)
	{
		m[j] = msg[i];
	}
	
	// Append a single '1' bit
	m[j++] = 0x80;  //1000 0000
	
	// Append K '0' bits, where k is the minimum number >= 0 such that L + 1 + K + 64 is a multiple of 512
	if(j > 56)
	{
		sha256_transform(ctx, m);
		memset(m, 0, sizeof(m));
		// printf("true\n");
	}
	
	// Append L as a 64-bit bug-endian integer, making the total post-processed length a multiple of 512 bits
	unsigned long long L = len * 8;  //bits
	m[63] = L;
	m[62] = L >> 8;
	m[61] = L >> 16;
	m[60] = L >> 24;
	m[59] = L >> 32;
	m[58] = L >> 40;
	m[57] = L >> 48;
	m[56] = L >> 56;
	sha256_transform(ctx, m);
	
	// Produce the final hash value (little-endian to big-endian)
	// Swap 1st & 4th, 2nd & 3rd byte for each word
	_swap(ctx->b[0], ctx->b[3]);
    _swap(ctx->b[1], ctx->b[2]);

    _swap(ctx->b[4], ctx->b[7]);
    _swap(ctx->b[5], ctx->b[6]);

    _swap(ctx->b[8], ctx->b[11]);
    _swap(ctx->b[9], ctx->b[10]);

    _swap(ctx->b[12], ctx->b[15]);
    _swap(ctx->b[13], ctx->b[14]);

    _swap(ctx->b[16], ctx->b[19]);
    _swap(ctx->b[17], ctx->b[18]);

    _swap(ctx->b[20], ctx->b[23]);
    _swap(ctx->b[21], ctx->b[22]);

    _swap(ctx->b[24], ctx->b[27]);
    _swap(ctx->b[25], ctx->b[26]);

    _swap(ctx->b[28], ctx->b[31]);
    _swap(ctx->b[29], ctx->b[30]);
}

// host version
void h_sha256_transform(SHA256 *ctx, const BYTE *msg)
{
	WORD a, b, c, d, e, f, g, h;
	WORD i, j;
	
	// Create a 64-entry message schedule array w[0..63] of 32-bit words
	WORD w[64];
	// Copy chunk into first 16 words w[0..15] of the message schedule array
	for(i=0, j=0;i<16;++i, j+=4)
	{
		w[i] = (msg[j]<<24) | (msg[j+1]<<16) | (msg[j+2]<<8) | (msg[j+3]);
	}
	
	// Extend the first 16 words into the remaining 48 words w[16..63] of the message schedule array:
	for(i=16;i<64;++i)
	{
		WORD s0 = (_rotr(w[i-15], 7)) ^ (_rotr(w[i-15], 18)) ^ (w[i-15]>>3);
		WORD s1 = (_rotr(w[i-2], 17)) ^ (_rotr(w[i-2], 19))  ^ (w[i-2]>>10);
		w[i] = w[i-16] + s0 + w[i-7] + s1;
	}
	
	
	// Initialize working variables to current hash value
	a = ctx->h[0];
	b = ctx->h[1];
	c = ctx->h[2];
	d = ctx->h[3];
	e = ctx->h[4];
	f = ctx->h[5];
	g = ctx->h[6];
	h = ctx->h[7];
	
	// Compress function main loop:
	for(i=0;i<64;++i)
	{
		WORD S0 = (_rotr(a, 2)) ^ (_rotr(a, 13)) ^ (_rotr(a, 22));
		WORD S1 = (_rotr(e, 6)) ^ (_rotr(e, 11)) ^ (_rotr(e, 25));
		WORD ch = (e & f) ^ ((~e) & g);
		WORD maj = (a & b) ^ (a & c) ^ (b & c);
		WORD temp1 = h + S1 + ch + h_k[i] + w[i];
		WORD temp2 = S0 + maj;
		
		h = g;
		g = f;
		f = e;
		e = d + temp1;
		d = c;
		c = b;
		b = a;
		a = temp1 + temp2;
	}
	
	// Add the compressed chunk to the current hash value
	ctx->h[0] += a;
	ctx->h[1] += b;
	ctx->h[2] += c;
	ctx->h[3] += d;
	ctx->h[4] += e;
	ctx->h[5] += f;
	ctx->h[6] += g;
	ctx->h[7] += h;
	
}

void h_sha256(SHA256 *ctx, const BYTE *msg, size_t len)
{
	// Initialize hash values:
	// (first 32 bits of the fractional parts of the square roots of the first 8 primes 2..19):
	ctx->h[0] = 0x6a09e667;
	ctx->h[1] = 0xbb67ae85;
	ctx->h[2] = 0x3c6ef372;
	ctx->h[3] = 0xa54ff53a;
	ctx->h[4] = 0x510e527f;
	ctx->h[5] = 0x9b05688c;
	ctx->h[6] = 0x1f83d9ab;
	ctx->h[7] = 0x5be0cd19;
	
	
	WORD i, j;
	size_t remain = len % 64;
	size_t total_len = len - remain;
	
	// Process the message in successive 512-bit chunks
	// For each chunk:
	for(i=0;i<total_len;i+=64)
	{
		h_sha256_transform(ctx, &msg[i]);
	}
	
	// Process remain data
	BYTE m[64] = {};
	for(i=total_len, j=0;i<len;++i, ++j)
	{
		m[j] = msg[i];
	}
	
	// Append a single '1' bit
	m[j++] = 0x80;  //1000 0000
	
	// Append K '0' bits, where k is the minimum number >= 0 such that L + 1 + K + 64 is a multiple of 512
	if(j > 56)
	{
		h_sha256_transform(ctx, m);
		memset(m, 0, sizeof(m));
		printf("true\n");
	}
	
	// Append L as a 64-bit bug-endian integer, making the total post-processed length a multiple of 512 bits
	unsigned long long L = len * 8;  //bits
	m[63] = L;
	m[62] = L >> 8;
	m[61] = L >> 16;
	m[60] = L >> 24;
	m[59] = L >> 32;
	m[58] = L >> 40;
	m[57] = L >> 48;
	m[56] = L >> 56;
	h_sha256_transform(ctx, m);
	
	// Produce the final hash value (little-endian to big-endian)
	// Swap 1st & 4th, 2nd & 3rd byte for each word
	for(i=0;i<32;i+=4)
	{
        _swap(ctx->b[i], ctx->b[i+3]);
        _swap(ctx->b[i+1], ctx->b[i+2]);
	}
}

////////////////////////   Utils   ///////////////////////

//convert one hex-codec char to binary
unsigned char decode(unsigned char c)
{
    switch(c)
    {
        case 'a':
            return 0x0a;
        case 'b':
            return 0x0b;
        case 'c':
            return 0x0c;
        case 'd':
            return 0x0d;
        case 'e':
            return 0x0e;
        case 'f':
            return 0x0f;
        case '0' ... '9':
            return c-'0';
    }
}


// convert hex string to binary
//
// in: input string
// string_len: the length of the input string
//      '\0' is not included in string_len!!!
// out: output bytes array
void convert_string_to_little_endian_bytes(unsigned char* out, char *in, size_t string_len)
{
    assert(string_len % 2 == 0);

    size_t s = 0;
    size_t b = string_len/2-1;

    for(s, b; s < string_len; s+=2, --b)
    {
        out[b] = (unsigned char)(decode(in[s])<<4) + decode(in[s+1]);
    }
}

// print out binary array (from highest value) in the hex format
void print_hex(unsigned char* hex, size_t len)
{
    for(int i=0;i<len;++i)
    {
        printf("%02x", hex[i]);
    }
    printf("\n");
}


// print out binar array (from lowest value) in the hex format
void print_hex_inverse(unsigned char* hex, size_t len)
{
    for(int i=len-1;i>=0;--i)
    {
        printf("%02x", hex[i]);
    }
    printf("\n");
}

// __device__ int little_endian_bit_comparison(const unsigned char *a, const unsigned char *b)
// {
//     const unsigned int *a_int = reinterpret_cast<const unsigned int*>(a);
//     const unsigned int *b_int = reinterpret_cast<const unsigned int*>(b);
//     // compared from lowest bit
//     int result = 0;
//     result = (result << 1) + (a_int[7] > b_int[7]) - (a_int[7] < b_int[7]);
//     result = (result << 1) + (a_int[6] > b_int[6]) - (a_int[6] < b_int[6]);
//     result = (result << 1) + (a_int[5] > b_int[5]) - (a_int[5] < b_int[5]);
//     result = (result << 1) + (a_int[4] > b_int[4]) - (a_int[4] < b_int[4]);
//     result = (result << 1) + (a_int[3] > b_int[3]) - (a_int[3] < b_int[3]);
//     result = (result << 1) + (a_int[2] > b_int[2]) - (a_int[2] < b_int[2]);
//     result = (result << 1) + (a_int[1] > b_int[1]) - (a_int[1] < b_int[1]);
//     result = (result << 1) + (a_int[0] > b_int[0]) - (a_int[0] < b_int[0]);
//     return result;
// }

void getline(char *str, size_t len, FILE *fp)
{

    int i=0;
    while( i<len && (str[i] = fgetc(fp)) != EOF && str[i++] != '\n');
    str[len-1] = '\0';
}

////////////////////////   Hash   ///////////////////////

__device__ void double_sha256(SharedData *data)
{
    sha256(&data->tmp, (BYTE*)&data->block, sizeof(HashBlock));
    sha256(&data->sha256_ctx, (BYTE*)&data->tmp, sizeof(SHA256));
}
void h_double_sha256(SHA256 *sha256_ctx, unsigned char *bytes, size_t len)
{
    SHA256 tmp;
    h_sha256(&tmp, (BYTE*)bytes, len);
    h_sha256(sha256_ctx, (BYTE*)&tmp, sizeof(tmp));
}

////////////////////   Find Nonce   /////////////////////


__global__ void find_nonce(__restrict__ HashBlock *block, unsigned char* __restrict__ target, unsigned int *solution) {
    __shared__ SharedData d_data[BLOCK_SIZE];

    d_data[threadIdx.x].block = *block;

    unsigned int nonce = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int stride = gridDim.x * blockDim.x;

    for (; 0xffffffff - nonce >= stride && !solution[0]; nonce += stride) {
        d_data[threadIdx.x].block.nonce = nonce;

        // Compute double SHA-256
        double_sha256(&d_data[threadIdx.x]);

        // Check if the hash is less than the target
        const unsigned int *a_int = reinterpret_cast<const unsigned int*>(d_data[threadIdx.x].sha256_ctx.b);
        const unsigned int *b_int = reinterpret_cast<const unsigned int*>(target);
        // compared from lowest bit
        int result = 0;
        result = (result << 1) + (a_int[7] > b_int[7]) - (a_int[7] < b_int[7]);
        result = (result << 1) + (a_int[6] > b_int[6]) - (a_int[6] < b_int[6]);
        result = (result << 1) + (a_int[5] > b_int[5]) - (a_int[5] < b_int[5]);
        result = (result << 1) + (a_int[4] > b_int[4]) - (a_int[4] < b_int[4]);
        result = (result << 1) + (a_int[3] > b_int[3]) - (a_int[3] < b_int[3]);
        result = (result << 1) + (a_int[2] > b_int[2]) - (a_int[2] < b_int[2]);
        result = (result << 1) + (a_int[1] > b_int[1]) - (a_int[1] < b_int[1]);
        result = (result << 1) + (a_int[0] > b_int[0]) - (a_int[0] < b_int[0]);

        // Write the solution and signal that a valid nonce is found
        solution[(result >= 0)] = nonce;
    }
}

////////////////////   Merkle Root   /////////////////////


// calculate merkle root from several merkle branches
// root: output hash will store here (little-endian)
// branch: merkle branch  (big-endian)
// count: total number of merkle branch
void calc_merkle_root(unsigned char *root, int count, char **branch)
{
    size_t total_count = count; // merkle branch
    unsigned char *raw_list = new unsigned char[(total_count+1)*32];
    unsigned char **list = new unsigned char*[total_count+1];

    // copy each branch to the list
    for(int i=0;i<total_count; ++i)
    {
        list[i] = raw_list + i * 32;
        //convert hex string to bytes array and store them into the list
        convert_string_to_little_endian_bytes(list[i], branch[i], 64);
    }

    list[total_count] = raw_list + total_count*32;


    // calculate merkle root
    while(total_count > 1)
    {
        
        // hash each pair
        int i, j;

        if(total_count % 2 == 1)  //odd, 
        {
            memcpy(list[total_count], list[total_count-1], 32);
        }

        for(i=0, j=0;i<total_count;i+=2, ++j)
        {
            // this part is slightly tricky,
            //   because of the implementation of the double_sha256,
            //   we can avoid the memory begin overwritten during our sha256d calculation
            // double_sha:
            //     tmp = hash(list[0]+list[1])
            //     list[0] = hash(tmp)
            h_double_sha256((SHA256*)list[j], list[i], 64);
        }

        total_count = j;
    }

    memcpy(root, list[0], 32);

    delete[] raw_list;
    delete[] list;
}


void solve(FILE *fin, FILE *fout)
{

    // **** read data *****
    char version[9];
    char prevhash[65];
    char ntime[9];
    char nbits[9];
    int tx;
    char *raw_merkle_branch;
    char **merkle_branch;

    getline(version, 9, fin);
    getline(prevhash, 65, fin);
    getline(ntime, 9, fin);
    getline(nbits, 9, fin);
    fscanf(fin, "%d\n", &tx);
    printf("start hashing");

    raw_merkle_branch = new char [tx * 65];
    merkle_branch = new char *[tx];
    for(int i=0;i<tx;++i)
    {
        merkle_branch[i] = raw_merkle_branch + i * 65;
        getline(merkle_branch[i], 65, fin);
        merkle_branch[i][64] = '\0';
    }

    // **** calculate merkle root ****

    unsigned char merkle_root[32];
    calc_merkle_root(merkle_root, tx, merkle_branch);

    // fprintf(stderr, "merkle root(little): ");
    // print_hex(merkle_root, 32);
    // fprintf(stderr, "\n");

    // fprintf(stderr, "merkle root(big):    ");
    // print_hex_inverse(merkle_root, 32);
    // fprintf(stderr, "\n");

    // **** solve block ****
    // fprintf(stderr, "Block info (big): \n");
    // fprintf(stderr, "  version:  %s\n", version);
    // fprintf(stderr, "  pervhash: %s\n", prevhash);
    // fprintf(stderr, "  merkleroot: "); print_hex_inverse(merkle_root, 32); fprintf(stderr, "\n");
    // fprintf(stderr, "  nbits:    %s\n", nbits);
    // fprintf(stderr, "  ntime:    %s\n", ntime);
    // fprintf(stderr, "  nonce:    ???\n\n");

    HashBlock block;

    // convert to byte array in little-endian
    convert_string_to_little_endian_bytes((unsigned char *)&block.version, version, 8);
    convert_string_to_little_endian_bytes(block.prevhash,                  prevhash,    64);
    memcpy(block.merkle_root, merkle_root, 32);
    convert_string_to_little_endian_bytes((unsigned char *)&block.nbits,   nbits,     8);
    convert_string_to_little_endian_bytes((unsigned char *)&block.ntime,   ntime,     8);
    block.nonce = 0;
    
    
    // ********** calculate target value *********
    // calculate target value from encoded difficulty which is encoded on "nbits"
    unsigned int exp = block.nbits >> 24;
    unsigned int mant = block.nbits & 0xffffff;
    unsigned char target_hex[32] = {};
    
    unsigned int shift = 8 * (exp - 3);
    unsigned int sb = shift / 8;
    unsigned int rb = shift % 8;
    
    // little-endian
    target_hex[sb    ] = (mant << rb);
    target_hex[sb + 1] = (mant >> (8-rb));
    target_hex[sb + 2] = (mant >> (16-rb));
    target_hex[sb + 3] = (mant >> (24-rb));
    
    
    // printf("Target value (big): ");
    // print_hex_inverse(target_hex, 32);
    // printf("\n");
    // fflush(stdout);


    // fprintf(stderr, "start to find nonce\n");
    // ********** find nonce **************
    HashBlock *d_block;
    unsigned char *d_target;
    unsigned int *d_solution;
    unsigned int h_solution[2] = {};
    unsigned char h_found_flag = 0;
    unsigned char *d_found_flag;
    
    hipError_t err;
    
    // Allocate memory on the device
    hipMalloc(&d_block, sizeof(HashBlock));
    hipMalloc(&d_target, 32 * sizeof(unsigned char));
    hipMalloc(&d_solution, sizeof(unsigned int));
    // hipMalloc(&d_found_flag, sizeof(unsigned char));

    err = hipGetLastError();
    if (err != hipSuccess){
        fprintf(stderr, "hipMalloc error: %s\n", hipGetErrorString(err));
    }

    // Copy data to the device
    // hipMemcpy(d_found_flag, &h_found_flag, sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(d_solution, h_solution, 2 * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(d_block, &block, sizeof(HashBlock), hipMemcpyHostToDevice);
    hipMemcpy(d_target, target_hex, 32 * sizeof(unsigned char), hipMemcpyHostToDevice);

    err = hipGetLastError();
    if (err != hipSuccess){
        fprintf(stderr, "hipMemcpy error: %s\n", hipGetErrorString(err));
    }

    // Set the cache configuration for the kernel
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(find_nonce), hipFuncCachePreferShared);

    auto start_kernel = std::chrono::high_resolution_clock::now();
    // Launch kernel
    int threads_per_block = 128;
    int blocks_per_grid = 2560; // Adjust based on your GPU
    find_nonce<<<blocks_per_grid, threads_per_block>>>(d_block, d_target, d_solution);

    err = hipGetLastError();
    if (err != hipSuccess){
        fprintf(stderr, "find_nonce error: %s\n", hipGetErrorString(err));
    }

    hipDeviceSynchronize();

    err = hipGetLastError();
    if (err != hipSuccess){
        fprintf(stderr, "DeviceSynchronize error: %s\n", hipGetErrorString(err));
    }
    auto end_kernel = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> kernel_time = end_kernel - start_kernel;
    std::cout << " Kernel Time: " << kernel_time.count() << " s" << std::endl;

    // Copy the result back to the host
    hipMemcpy(&h_solution, d_solution, sizeof(unsigned int), hipMemcpyDeviceToHost);

    err = hipGetLastError();
    if (err != hipSuccess){
        fprintf(stderr, "hipMemcpy error: %s\n", hipGetErrorString(err));
    }
    
    // SHA256 sha256_ctx;
    
    // for(block.nonce=0x00000000; block.nonce<=0xf;++block.nonce)
    // {   
    //     //sha256d
    //     h_double_sha256(&sha256_ctx, (unsigned char*)&block, sizeof(block));
    //     print_hex(sha256_ctx.b, 32);
        
    //     if(little_endian_bit_comparison(sha256_ctx.b, target_hex, 32) < 0)  // sha256_ctx < target_hex
    //     {
    //         printf("Found Solution!!\n");
    //         printf("hash #%10u (big): ", block.nonce);
    //         print_hex_inverse(sha256_ctx.b, 32);
    //         printf("\n\n");

    //         break;
    //     }
    // }
    

    // print result

    //little-endian
    // printf("hash(little): ");
    // print_hex(sha256_ctx.b, 32);
    // printf("\n");

    // //big-endian
    // printf("hash(big):    ");
    // print_hex_inverse(sha256_ctx.b, 32);
    // printf("\n\n");

    fprintf(stderr, "nonce: %u\n", h_solution[0]);

    for(int i=0;i<4;++i)
    {
        fprintf(fout, "%02x", ((unsigned char*)&h_solution)[i]);
    }
    fprintf(fout, "\n");

    // for(int i=0;i<4;++i)
    // {
    //     fprintf(fout, "%02x", ((unsigned char*)&block.nonce)[i]);
    // }
    // fprintf(fout, "\n");
    

    delete[] merkle_branch;
    delete[] raw_merkle_branch;
}

int main(int argc, char **argv)
{
    if (argc != 3) {
        fprintf(stderr, "usage: cuda_miner <in> <out>\n");
    }
    FILE *fin = fopen(argv[1], "r");
    FILE *fout = fopen(argv[2], "w");

    int totalblock;

    fscanf(fin, "%d\n", &totalblock);
    fprintf(fout, "%d\n", totalblock);

    for(int i=0;i<totalblock;++i)
    {
        solve(fin, fout);
    }

    return 0;
}

